#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <chrono>
#include <stdexcept>
#include <string>

#include "hip/hip_runtime.h"

using namespace std;



std::vector<std::vector<float>> readMatrix(const std::string& filename, size_t& rows, size_t& cols) 
{
    std::ifstream file(filename);
    if (!file.is_open())
        throw std::runtime_error("Cannot open file: " + filename);

    std::vector<std::vector<float>> matrix;
    rows = 0;
    std::string line;

    while (std::getline(file, line)) 
    {
        std::stringstream ss(line);
        std::vector<float> row;
        float value;

        while (ss >> value) 
        {
            row.push_back(value);
        }

        if (!row.empty()) 
        {
            matrix.push_back(row);
            cols = std::max(cols, row.size());
            rows++;
        }
    }

    file.close();

    if (rows == 0)
        throw std::runtime_error("File is empty or contains no valid data");

    for (auto& row : matrix) 
    {
        row.resize(cols, 0.0f);
    }

    return matrix;
}


__global__ void matrixMultiplyKernel(float *A, float *B, float *C, int A_rows, int A_cols, int B_cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < A_rows && col < B_cols)
    {
        float sum = 0.0f;
        for (int i = 0; i < A_cols; ++i)
        {
            sum += A[row * A_cols + i] * B[i * B_cols + col];
        }
        C[row * B_cols + col] = sum;
    }
}


vector<vector<float>> mulMatricesCUDA(const vector<vector<float>> &A, const vector<vector<float>> &B)
{
    int A_rows = A.size(), A_cols = A[0].size(), B_cols = B[0].size();

    vector<float> h_A(A_rows * A_cols);
    vector<float> h_B(A_cols * B_cols);
    vector<float> h_C(A_rows * B_cols);

    for (int i = 0; i < A_rows; ++i)
        for (int j = 0; j < A_cols; ++j)
            h_A[i * A_cols + j] = A[i][j];

    for (int i = 0; i < A_cols; ++i)
        for (int j = 0; j < B_cols; ++j)
            h_B[i * B_cols + j] = B[i][j];

    float *d_A, *d_B, *d_C;
    size_t size_A = h_A.size() * sizeof(float);
    size_t size_B = h_B.size() * sizeof(float);
    size_t size_C = h_C.size() * sizeof(float);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A.data(), size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size_B, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((B_cols + 15) / 16, (A_rows + 15) / 16);

    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, A_rows, A_cols, B_cols);
    hipDeviceSynchronize();

    hipMemcpy(h_C.data(), d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    vector<vector<float>> result(A_rows, vector<float>(B_cols));
    for (int i = 0; i < A_rows; ++i)
        for (int j = 0; j < B_cols; ++j)
            result[i][j] = h_C[i * B_cols + j];

    return result;
}


void writeMatrix(const std::string& filename, const std::vector<std::vector<float>>& matrix) 
{
    std::ofstream file(filename);
    if (!file.is_open())
        throw std::runtime_error("Cannot open file for writing: " + filename);

    for (const auto& row : matrix) 
    {
        for (float val : row)
            file << val << " ";
        file << "\n";
    }
    file.close();
}


int main()
{
    try
    {
        long total_time = 0;
        size_t rowsA, colsA = 0, rowsB, colsB = 0;

        std::vector<std::pair<std::string, std::string>> files = {
            {"../../Matrix_1/matrix1_10.txt", "../../Matrix_2/matrix2_10.txt"},
            {"../../Matrix_1/matrix1_20.txt", "../../Matrix_2/matrix2_20.txt"},
            {"../../Matrix_1/matrix1_30.txt", "../../Matrix_2/matrix2_30.txt"},
            {"../../Matrix_1/matrix1_40.txt", "../../Matrix_2/matrix2_40.txt"},
            {"../../Matrix_1/matrix1_50.txt", "../../Matrix_2/matrix2_50.txt"},
            {"../../Matrix_1/matrix1_60.txt", "../../Matrix_2/matrix2_60.txt"},
            {"../../Matrix_1/matrix1_70.txt", "../../Matrix_2/matrix2_70.txt"},
            {"../../Matrix_1/matrix1_80.txt", "../../Matrix_2/matrix2_80.txt"},
            {"../../Matrix_1/matrix1_90.txt", "../../Matrix_2/matrix2_90.txt"},
            {"../../Matrix_1/matrix1_100.txt", "../../Matrix_2/matrix2_100.txt"},
            {"../../Matrix_1/matrix1_1000.txt", "../../Matrix_2/matrix2_1000.txt"}
        };

        for (size_t i = 0; i < files.size(); i++)
        {
            std::string fileA = files[i].first;
            std::string fileB = files[i].second;
            std::string resultFile = "../../Output/output_" + std::to_string((i + 1) * 10) + ".txt";

            auto matrixA = readMatrix(fileA, rowsA, colsA);
            auto matrixB = readMatrix(fileB, rowsB, colsB);

            if (colsA != rowsB)
                throw std::runtime_error("Matrices cannot be multiplied: colsA != rowsB");

            auto start = std::chrono::high_resolution_clock::now();
            auto result = mulMatricesCUDA(matrixA, matrixB);
            auto end = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
            total_time += duration;

            writeMatrix(resultFile, result);

            std::cout << "Lead time for " << fileA << " and " << fileB << ": " << duration << " us" << std::endl;
            std::cout << "Task volume: " << rowsA << " * " << colsB << std::endl;
            std::cout << "The result is written to the file: " << resultFile << std::endl;
        }

        std::cout << "Total lead time: " << total_time << " us" << std::endl;
    }
    catch(const std::exception& ex)
    {
        std::cerr << "Error: " << ex.what() << std::endl;
        return -1; 
    }
    return 0;
}